#include "hip/hip_runtime.h"
#include <cstdio>

#ifdef STANDALONE
#include <cstdlib>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>
#include <assert.h>
#else
#include <SxComplex.h>
#include "SxGpu.h"
#endif

#include <hip/hip_runtime.h>
#include <hipblas.h>

#undef CUDA_DEBUG
//#define CUDA_DEBUG
#ifdef CUDA_DEBUG
#define CU_CHECK( CU_CALL )                                \
{                                                           \
   CU_CALL;                                                 \
   hipDeviceSynchronize();                                 \
   hipError_t status = hipGetLastError();                 \
   if (status != hipSuccess)                               \
   {                                                        \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);  \
      /*exit(1);*/                                          \
   }                                                        \
}
#else
#define CU_CHECK( CU_CALL )                                \
{                                                           \
   CU_CALL;                                                 \
}
#endif


namespace gpu
{
   int dev = 0;
   namespace mmm3vars
   {
      bool initialized = false;
      // --- CUDA device pointers
      hipDoubleComplex *A_d, *B_d, *R_d, *BC_d;
      double *C_d;
      // --- CUDA grid parameters
      dim3 block;
      dim3 grid;
      // --- CUDA hardware parameters
      int warpsize;
      int maxThreadsPerBlock;
      // --- handle for CUBLAS ZGEMM
      hipblasHandle_t cublasHandle;
      // --- factors for CUBLAS ZGEMM
      const hipDoubleComplex alpha = {1.0, 0.0};
      const hipDoubleComplex beta  = {0.0, 0.0};

      __global__
      void knl_mmm3_calcBC(int nb, int nc, int nd,
            hipDoubleComplex *B, double *C, hipDoubleComplex *BC)
      {
         int id = blockIdx.x*blockDim.x + threadIdx.x;
         int ib = blockIdx.y*blockDim.y + threadIdx.y;
         int ic = blockIdx.z*blockDim.z + threadIdx.z;

         if ( (id < nd) && (ic < nc) && (ib < nb) )
         {
            int idxb = id + ib*nd;
            int idxc = id + ic*nd;
            int idxr = id + ic*nd + ib*nd*nc;
            BC[idxr] = make_hipDoubleComplex( hipCreal(B[idxb]) * C[idxc],
                                             hipCimag(B[idxb]) * C[idxc] );
         }
      }
   } // namespace mmm3vars

   void mmm3(int Na, int Nb, int Nc, int Nd,
              const hipDoubleComplex *A, const hipDoubleComplex *B, const double *C, hipDoubleComplex *R,
              int rldc, int rldbc)
   {
      using namespace mmm3vars;
      if (!initialized)
      {
         hipSetDevice(dev);
         hipDeviceProp_t prop;
         CU_CHECK( hipGetDeviceProperties(&prop, dev) );
         warpsize = prop.warpSize;
         maxThreadsPerBlock = prop.maxThreadsPerBlock;

         // --- set up grid configuration for BC calculation
         int blockSize = warpsize;
         block.x = blockSize;
         block.y = blockSize;
         block.z = 1;
         grid.x  = (unsigned)ceil((Nd)/(double)(block.x));
         grid.y  = (unsigned)ceil((Nb)/(double)(block.y));
         grid.z  = (unsigned)ceil((Nc)/(double)(block.z));

         printf("MMM3 CUDA launch configuration : block(%d,%d,%d), grid(%d,%d,%d)\n",
                block.x,block.y,block.z, grid.x,grid.y,grid.z);

         CU_CHECK( hipMalloc( (void**)&A_d,     Na*Nd*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&B_d,     Nb*Nd*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&C_d,     Nc*Nd*sizeof(double)) );
         CU_CHECK( hipMalloc( (void**)&BC_d, Nd*Nc*Nb*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&R_d,  Na*rldbc*sizeof(hipDoubleComplex)) );

         hipblasCreate(&cublasHandle);

         initialized = true;
      }

      {
         CU_CHECK( hipMemcpy( A_d, A, Na*Nd*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
         CU_CHECK( hipMemcpy( B_d, B, Nb*Nd*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
         CU_CHECK( hipMemcpy( C_d, C, Nc*Nd*sizeof(double), hipMemcpyHostToDevice) );
      }

      // --- STEP 1 : calculate BC[d,c,b] = B[d,b] * C[d,c]
      {
         knl_mmm3_calcBC<<<grid,block>>>(Nb, Nc, Nd, B_d, C_d, BC_d);
         CU_CHECK();
      }

      // --- STEP 2 : calculate R[c,b,a] = sum_d BC[d,c,b] A[d,a] using zgemm
      {
         hipblasZgemm       // --- documentation for input parameters, cf CUBLAS guide
         (
            cublasHandle,  // input handle to the cuBLAS library context
            HIPBLAS_OP_T,   // input operation op(A) that is non- or (conj.) transpose
            HIPBLAS_OP_N,   // input operation op(B) that is non- or (conj.) transpose
            rldbc,         // m -- input number of rows of matrix op(A) and C
            Na,            // n -- input number of columns of matrix op(B) and C
            Nd,            // k -- input number of columns of op(A) and rows of op(B)
            &alpha,        // scalar used for multiplication
            BC_d,          // A -- array of dimensions lda x k with lda>=max(1,m) if transa == HIPBLAS_OP_N and lda x m with lda>=max(1,k) otherwise
            Nd,            // lda -- input leading dimension of two-dimensional array used to store the matrix A
            A_d,           // B -- array of dimension ldb x n with ldb>=max(1,k) if transa == HIPBLAS_OP_N and ldb x k with ldb>=max(1,n) otherwise
            Nd,            // ldb -- input leading dimension of two-dimensional array used to store the matrix B
            &beta,         // scalar used for multiplication
            R_d,           // C -- array of dimensions ldc x n with ldc>=max(1,m)
            rldbc          // ldc -- input leading dimension of a two-dimensional array used to store the matrix C
         );
         CU_CHECK();
      }

      {
         CU_CHECK( hipMemcpy( R, R_d, Na*rldbc*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
      }
   } // mmm3


   namespace mm3mvars
   {
      bool initialized = false;
      // --- CUDA device pointers
      hipDoubleComplex *B_d, *BC_d, *X_d, *R_d;
      double *C_d;
      // --- CUDA grid parameters
      dim3 block;
      dim3 grid;
      // --- CUDA hardware parameters
      int warpsize;
      int maxThreadsPerBlock;
      // --- handle for CUBLAS ZGEMM
      hipblasHandle_t cublasHandle;
      // --- factors for CUBLAS ZGEMM
      const hipDoubleComplex alpha = {1.0, 0.0};
      const hipDoubleComplex beta  = {0.0, 0.0};

      __global__
      void knl_mm3m_calcBC(int nb, int nc, int nd,
            hipDoubleComplex *B, double *C, hipDoubleComplex *BC)
      {
         int id = blockIdx.x*blockDim.x + threadIdx.x;
         int ib = blockIdx.y*blockDim.y + threadIdx.y;
         int ic = blockIdx.z*blockDim.z + threadIdx.z;

         if ( (id < nd) && (ic < nc) && (ib < nb) )
         {
            int idxb = id + ib*nd;
            int idxc = id + ic*nd;
            int idxr = id + ic*nd + ib*nd*nc;
            BC[idxr] = make_hipDoubleComplex( hipCreal(B[idxb]) * C[idxc],
                                             hipCimag(B[idxb]) * C[idxc] );
         }
      }
   } // namespace mm3mvars


   void mm3m(int Na, int Nb, int Nc, int Nd,
              const hipDoubleComplex *B, const double *C, const hipDoubleComplex *X, hipDoubleComplex *R,
              int xldc, int xldbc)
   {
      using namespace mm3mvars;
      if (!initialized)
      {
         hipSetDevice(dev);
         hipDeviceProp_t prop;
         CU_CHECK( hipGetDeviceProperties(&prop, dev) );
         warpsize = prop.warpSize;
         maxThreadsPerBlock = prop.maxThreadsPerBlock;

         // --- set up grid configuration for BC calculation
         int blockSize = warpsize;
         block.x = blockSize;
         block.y = blockSize;
         block.z = 1;
         grid.x  = (unsigned)ceil((Nd)/(double)(block.x));
         grid.y  = (unsigned)ceil((Nb)/(double)(block.y));
         grid.z  = (unsigned)ceil((Nc)/(double)(block.z));

         printf("MM3M CUDA launch configuration : block(%d,%d,%d), grid(%d,%d,%d)\n",
                block.x,block.y,block.z, grid.x,grid.y,grid.z);

         CU_CHECK( hipMalloc( (void**)&B_d,     Nb*Nd*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&C_d,     Nc*Nd*sizeof(double)) );
         CU_CHECK( hipMalloc( (void**)&BC_d, Nd*Nc*Nb*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&X_d,  Na*xldbc*sizeof(hipDoubleComplex)) );
         CU_CHECK( hipMalloc( (void**)&R_d,     Na*Nd*sizeof(hipDoubleComplex)) );

         hipblasCreate(&cublasHandle);

         initialized = true;
      }

      {
         CU_CHECK( hipMemcpy( B_d, B,    Nb*Nd*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
         CU_CHECK( hipMemcpy( C_d, C,    Nc*Nd*sizeof(double),         hipMemcpyHostToDevice) );
         CU_CHECK( hipMemcpy( X_d, X, Na*xldbc*sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
      }

      // --- STEP 1 : calculate BC[d,c,b] = B[d,b] * C[d,c]
      {
         knl_mm3m_calcBC<<<grid,block>>>(Nb, Nc, Nd, B_d, C_d, BC_d);
         CU_CHECK();
      }

      // --- STEP 2 : calculate R[d,a] = sum_{b,c} BC[d,c,b] X[c,b,a] using zgemm
      {
         hipblasZgemm       // --- documentation for input parameters, cf CUBLAS guide
         (
            cublasHandle,  // input handle to the cuBLAS library context
            HIPBLAS_OP_N,   // input operation op(A) that is non- or (conj.) transpose
            HIPBLAS_OP_N,   // input operation op(B) that is non- or (conj.) transpose
            Nd,            // m -- input number of rows of matrix op(A) and C
            Na,            // n -- input number of columns of matrix op(B) and C
            xldbc,         // k -- input number of columns of op(A) and rows of op(B)
            &alpha,        // scalar used for multiplication
            BC_d,          // A -- array of dimensions lda x k with lda>=max(1,m) if transa == HIPBLAS_OP_N and lda x m with lda>=max(1,k) otherwise
            Nd,            // lda -- input leading dimension of two-dimensional array used to store the matrix A
            X_d,           // B -- array of dimension ldb x n with ldb>=max(1,k) if transa == HIPBLAS_OP_N and ldb x k with ldb>=max(1,n) otherwise
            xldbc,         // ldb -- input leading dimension of two-dimensional array used to store the matrix B
            &beta,         // scalar used for multiplication
            R_d,           // C -- array of dimensions ldc x n with ldc>=max(1,m)
            Nd             // ldc -- input leading dimension of a two-dimensional array used to store the matrix C
         );
         CU_CHECK();
      }

      {
         CU_CHECK( hipMemcpy( R, R_d, Na*Nd*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
      }
   } // mm3m

   void free()
   {
      {
         using namespace mmm3vars;
         if (initialized)
         {
            CU_CHECK( hipFree(A_d)  );
            CU_CHECK( hipFree(B_d)  );
            CU_CHECK( hipFree(C_d)  );
            CU_CHECK( hipFree(BC_d) );
            CU_CHECK( hipFree(R_d)  );
            hipblasDestroy(cublasHandle);
            initialized = false;
         }
      }
      {
         using namespace mm3mvars;
         if (initialized)
         {
            CU_CHECK( hipFree(B_d)  );
            CU_CHECK( hipFree(C_d)  );
            CU_CHECK( hipFree(BC_d) );
            CU_CHECK( hipFree(X_d)  );
            CU_CHECK( hipFree(R_d)  );
            hipblasDestroy(cublasHandle);
            initialized = false;
         }
      }
   }

} // namespace gpu





#ifdef STANDALONE

using namespace std;

typedef struct {
   double re;
   double im;
} CMPLX;

namespace cpu {
   #if defined __GNUG__
      #define RESTRICT __restrict__
   #else
      #define RESTRICT
   #endif

   inline void mmm3_d_loop(const int Nd, CMPLX * RESTRICT x, CMPLX * RESTRICT y, double * RESTRICT z, CMPLX * RESTRICT r)
   {
      CMPLX sum = {0.0, 0.0};
      for (int d=0; d<Nd; ++d)
      {
         CMPLX xy;
         // --- X*Y
         xy.re   = x[d].re * y[d].re - x[d].im * y[d].im;
         xy.im   = x[d].re * y[d].im + x[d].im * y[d].re;
         // --- XY*Z
         sum.re += xy.re * z[d];
         sum.im += xy.im * z[d];
      }
      *r = sum;
   }

   void mmm3(int Na, int Nb, int Nc, int Nd,
              CMPLX * RESTRICT x, CMPLX * RESTRICT y, double * RESTRICT z, CMPLX * RESTRICT r)
   {
      int a, idxx, b, idxy, c, idxz, idxr;

#pragma omp parallel for \
   default(none) \
   private(a,idxx,b,idxy,c,idxz,idxr) \
   shared(Na,Nb,Nc,Nd,r,x,y,z)
      for (a=0; a<Na; ++a)
      {
         idxx = a*Nd;
         for (b=0; b<Nb; ++b)
         {
            idxy = b*Nd;
            for (c=0; c<Nc; ++c)
            {
               idxz = c*Nd;
               idxr = a*Nb*Nc + b*Nc + c;
               mmm3_d_loop(Nd, &x[idxx], &y[idxy], &z[idxz], &r[idxr]);
            }
         }
      }
   }

   inline CMPLX mm3m_3mul_add(const CMPLX &B, const double &C, const CMPLX &X)
   {
      CMPLX BC;
      // --- B*C
      BC.re = B.re * C;
      BC.im = B.im * C;
      // --- BC*X
      CMPLX R;
      R.re = BC.re * X.re - BC.im * X.im;
      R.im = BC.re * X.im + BC.im * X.re;
      // ---
      return R;
   }

   // --- naive mm3m CPU implementation
   void mm3m(int Na, int Nb, int Nc, int Nd,
            CMPLX * RESTRICT B, double * RESTRICT C, CMPLX * RESTRICT X, CMPLX * RESTRICT R)
   {
      int a, b, c, d;
      int idxb, idxc, idxx, idxr;
      CMPLX BCX;

#pragma omp parallel for \
   default(none) \
   private(a,b,c,d,idxb,idxc,idxx,idxr,BCX) \
   shared(Na,Nb,Nc,Nd,B,C,X,R)
      for (a=0; a<Na; ++a)
      {
         for (d=0; d<Nd; ++d)
         {
            idxr = a*Nd + d;
            R[idxr].re = 0.0;
            R[idxr].im = 0.0;
            //idxr *= 2;
            for (c=0; c<Nc; ++c)
            {
               idxc = c*Nd + d;
               //idxc *= 2;
               for (b=0; b<Nb; ++b)
               {
                  idxb = b*Nd + d;
                  //idxb *= 2;
                  idxx = a*Nb*Nc + b*Nc + c;
                  //idxx *= 2;
                  // --- definitely not optimized for speed
                  BCX = mm3m_3mul_add( B[idxb], C[idxc], X[idxx] );
                  R[idxr].re += BCX.re;
                  R[idxr].im += BCX.im;
               }
            }
         }
      }

   }


} // namespace cpu

   class stopwatch
   {
      struct timeval start;
      struct timeval stop;
      char label[128];
      double duration;
   public:
      stopwatch() {
         strcpy(label, "(no label)");
         gettimeofday(&start, NULL);
      }
      stopwatch(const char * label_) {
         strcpy(label, label_);
         gettimeofday(&start, NULL);
      }
      ~stopwatch() {
         gettimeofday(&stop, NULL);
         double dstart, dstop;
         dstart = (double) start.tv_sec + ((double) start.tv_usec)*1.e-6;
         dstop  = (double)  stop.tv_sec + ((double)  stop.tv_usec)*1.e-6;
         duration = dstop - dstart;
         printf("%s : %g s\n", label, duration);
      }

   };

   int main (int argc, char ** argv)
   {
      double *A, *B, *C, *X, *RC, *RG;

      int Na = 32;
      int Nb = 32;
      int Nc = 8;
      int Nd = 5000;

      // --- allowed relative difference between GPU and CPU results during the tests
      const double eps = 1.e-9;

      if (argc == 1)
      {
         /* use default parameters */
      }
      else if (argc == 5)
      {
         sscanf(argv[1], "%d", &Na);
         sscanf(argv[2], "%d", &Nb);
         sscanf(argv[3], "%d", &Nc);
         sscanf(argv[4], "%d", &Nd);
      }
      else
      {
         printf("Usage: %s Na Nb Nc Nd\n", argv[0]);
         return 1;
      }

      printf("parameters : Na=%d  Nb=%d  Nc=%d  Nd=%d\n",
                           Na,    Nb,    Nc,    Nd);

      // --- align memory to 64 bit boundaries (formerly to make MIC happy)
      assert(posix_memalign((void**)&A,  64,    Na*Nd*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&B,  64,    Nb*Nd*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&C,  64,    Nc*Nd*  sizeof(double)) == 0);
      assert(posix_memalign((void**)&RC, 64, Na*Nb*Nc*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&RG, 64, Na*Nb*Nc*2*sizeof(double)) == 0);

      srand(time(NULL));
      for (int i=0; i<Na*Nd*2; ++i) A[i] = double(rand()) / double(RAND_MAX);
      for (int i=0; i<Nb*Nd*2; ++i) B[i] = double(rand()) / double(RAND_MAX);
      for (int i=0; i<Nc*Nd  ; ++i) C[i] = double(rand()) / double(RAND_MAX);

      printf("\n");


      // (A) --- run MMM3 tests

      // (A1) --- run on the CPU
      {
         for (int j=0; j<Na*Nb*Nc*2; ++j) RC[j] = 0.;
         stopwatch sw("mmm3-cpu");
         cpu::mmm3(Na, Nb, Nc, Nd, (CMPLX*)A, (CMPLX*)B, C, (CMPLX*)RC);
      }
      printf("\n");

      // (A2) --- run on the GPU
      for (int i=0; i<2; ++i)
      {
         for (int j=0; j<Na*Nb*Nc*2; ++j) RG[j] = 0.;
         stopwatch sw("mmm3-gpu");
         gpu::mmm3 (Na, Nb, Nc, Nd, (hipDoubleComplex*)A, (hipDoubleComplex*)B, C, (hipDoubleComplex*)RG, Nc, Nb*Nc);
      }

      // (A3) --- compare CPU and accelerator results
      {
         for (int a=0; a<Na; ++a)
         {
            for (int b=0; b<Nb; ++b)
            {
               for (int c=0; c<Nc; ++c)
               {
                  int idxr = a*Nb*Nc + b*Nc + c;
                  idxr *= 2;
                  for (int i=0; i<2; ++i)
                  {
                     bool ok = fabs((RG[idxr] - RC[idxr])/RC[idxr]) < eps;
                     if (!ok)
                        printf("%d %d %d %d : %f %f\n", a, b, c, idxr, RG[idxr], RC[idxr]);
                     assert(ok);
                     ++idxr;
                  }
               }
            }
         }
      }
      printf("\nMMM3 : CPU and accelerator results match!\n");

      free(A);
      free(B);
      free(C);
      free(RC);
      free(RG);
      gpu::free();

      printf("\n");
      printf("\n");



      // (B) --- run MM3M tests

      assert(posix_memalign((void**)&B,  64,    Nb*Nd*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&C,  64,    Nc*Nd*  sizeof(double)) == 0);
      assert(posix_memalign((void**)&X,  64, Na*Nb*Nc*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&RC, 64,    Na*Nd*2*sizeof(double)) == 0);
      assert(posix_memalign((void**)&RG, 64,    Na*Nd*2*sizeof(double)) == 0);

      for (int i=0;    i<Nb*Nd*2; ++i) B[i] = double(rand()) / double(RAND_MAX);
      for (int i=0;    i<Nc*Nd  ; ++i) C[i] = double(rand()) / double(RAND_MAX);
      for (int i=0; i<Na*Nb*Nc*2; ++i) X[i] = double(rand()) / double(RAND_MAX);


      // (B1) --- run contraction on the CPU
      {
         for (int j=0; j<Na*Nd*2; ++j) RC[j] = 0.;
         stopwatch sw("mm3m-cpu");
         cpu::mm3m(Na, Nb, Nc, Nd, (CMPLX*)B, C, (CMPLX*)X, (CMPLX*)RC);
      }
      printf("\n");

      // (B2) --- run on the GPU
      for (int i=0; i<2; ++i)
      {
         for (int j=0; j<Na*Nd*2; ++j) RG[j] = 0.;
         stopwatch sw("mm3m-gpu");
         gpu::mm3m (Na, Nb, Nc, Nd, (hipDoubleComplex*)B, C, (hipDoubleComplex*)X, (hipDoubleComplex*)RG, Nc, Nb*Nc);
      }

      // (B3) --- compare CPU and accelerator results
      {
         for (int a=0; a<Na; ++a)
         {
            for (int d=0; d<Nd; ++d)
            {
               int idxr = a*Nd + d;
               idxr *= 2;
               for (int i=0; i<2; ++i)
               {
                  bool ok = fabs((RG[idxr] - RC[idxr])/RC[idxr]) < eps;
                  if (!ok)
                     printf("%d %d %d : %f %f\n", a, d, idxr, RG[idxr], RC[idxr]);
                  assert(ok);
                  ++idxr;
               }
            }
         }
      }
      printf("\nMM3M : CPU and accelerator results match!\n");

      free(B);
      free(C);
      free(X);
      free(RC);
      free(RG);
      gpu::free();

      return 0;
   }
#else // STANDALONE

void sx_gpu_gemmm3 (ssize_t Na, ssize_t Nb, ssize_t Nc, ssize_t Nd,
              const SxComplex16 *A,
              const SxComplex16 *B,
              const double *C,
              SxComplex16 *res, ssize_t rldc, ssize_t rldbc)
{
   gpu::mmm3 (int(Na), int(Nb), int(Nc), int(Nd),
         (hipDoubleComplex*)A, (hipDoubleComplex*)B, C,
         (hipDoubleComplex*)res,
         int(Nc), int(Nb*Nc));
}

void sx_gpu_gemm3m (ssize_t Na, ssize_t Nb, ssize_t Nc, ssize_t Nd,
              const SxComplex16 *B,
              const double *C,
              const SxComplex16 *X,
              SxComplex16 *res, ssize_t xldc, ssize_t xldbc)
{
   gpu::mm3m (int(Na), int(Nb), int(Nc), int(Nd),
         (hipDoubleComplex*)B, C, (hipDoubleComplex*)X,
         (hipDoubleComplex*)res,
         int(Nc), int(Nb*Nc));
}

void sx_gpu_gemmm_free ()
{
   gpu::free ();
}

void sx_gpu_set_device (int id)
{
   gpu::dev = id;
}

#endif // STANDALONE
